#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/common_layers.hpp"

namespace caffe {

template <typename Dtype>
void MultiSoftmaxWithLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
      Forward_cpu(bottom, top);
}

template <typename Dtype>
void MultiSoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
      Backward_cpu(top, propagate_down, bottom);     
}

INSTANTIATE_LAYER_GPU_FUNCS(MultiSoftmaxWithLossLayer);

}  // namespace caffe
